#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h>
#include <hiprand/hiprand_kernel.h>
#include <hip/driver_types.h>
#include <hiprand.h>
#include <unistd.h>
#include <stdlib.h>
#include <string.h>
#include <stdio.h>
#include <time.h>
#include <cstdio>
#include <hip/hip_runtime.h>

#include "support.h"
#include "hash_kernel.cu"
#include "nonce_kernel.cu"
#include "reduction_kernel.cu"

// to activate debug statements
#define DEBUG 1

// program constants
#define BLOCK_SIZE 1024
#define SEED       123

// solution constants
#define MAX     123123123
#define TARGET  20

//CITATIONS:
//chat GPT

// functions used
unsigned int generate_hash(unsigned int nonce, unsigned int index, unsigned int* transactions, unsigned int n_transactions);
void read_file(char* file, unsigned int* transactions, unsigned int n_transactions);
void err_check(hipError_t ret, char* msg, int exit_code);


/* Main ------------------ //
*   This is the main program.
*/
int main(int argc, char* argv[]) {

    // Catch console errors
    if (argc != 6) {
        printf("USE LIKE THIS: gpu_mining transactions.csv n_transactions trials out.csv time.csv\n");
        return EXIT_FAILURE;
    }


    // Output files
    FILE* output_file = fopen(argv[4], "w");
    FILE* time_file   = fopen(argv[5], "w");

    // Read in the transactions
    unsigned int n_transactions = strtoul(argv[2], NULL, 10);
    unsigned int* transactions = (unsigned int*)calloc(n_transactions, sizeof(unsigned int));
    read_file(argv[1], transactions, n_transactions);

    // get the number of trials
    unsigned int trials = strtoul(argv[3], NULL, 10);


    // -------- Start Mining ------------------------------------------------------- //
    // ----------------------------------------------------------------------------- //
    
    // Set timer and cuda error return
    Timer timer;
    startTime(&timer);
    hipError_t cuda_ret;

    // To use with kernels
    int num_blocks = ceil((float)trials / (float)BLOCK_SIZE);
    dim3 dimGrid(num_blocks, 1, 1);
    dim3 dimBlock(BLOCK_SIZE, 1, 1);


    // ------ Step 1: generate the nonce values ------ //

    // Allocate the nonce device memory
    unsigned int* device_nonce_array;
    cuda_ret = hipMalloc((void**)&device_nonce_array, trials * sizeof(unsigned int));
    err_check(cuda_ret, (char*)"Unable to allocate nonces to device memory!", 1);

    // Launch the nonce kernel
    nonce_kernel <<< dimGrid, dimBlock >>> (
        device_nonce_array, // put nonces into here
        trials,             // size of array
        MAX,                // to mod with
        SEED                // random seed
        );
    cuda_ret = hipDeviceSynchronize();
    err_check(cuda_ret, (char*)"Unable to launch nonce kernel!", 2);

    // Get nonces from device memory
    unsigned int* nonce_array = (unsigned int*)calloc(trials, sizeof(unsigned int));
    cuda_ret = hipMemcpy(nonce_array, device_nonce_array, trials * sizeof(unsigned int), hipMemcpyDeviceToHost);
    err_check(cuda_ret, (char*)"Unable to read nonce from device memory!", 3);


    // ------ Step 2: Generate the hash values ------ //

    // TODO Problem 1: perform this hash generation in the GPU
    // Hint: You need both nonces and transactions to compute a hash.
    unsigned int* device_hash_array;
    cuda_ret = hipMalloc((void**)&device_hash_array, trials * sizeof(unsigned int));
    err_check(cuda_ret, (char*)"Unable to allocate hash array to device memory!", 4);

    // Allocate transactions array in device memory
    unsigned int* device_transactions;
    cuda_ret = hipMalloc((void**)&device_transactions, n_transactions * sizeof(unsigned int));
    hipMemcpy(device_transactions, transactions, n_transactions* sizeof(unsigned int), hipMemcpyHostToDevice);


    // Launch the hash kernel
    hash_kernel <<< dimGrid, dimBlock >>> (
        device_hash_array,  // put hashes into here
        device_nonce_array, // use these nonces
        trials,             // size of arrays
        device_transactions,       // transactions to use in the hash
        n_transactions,     // number of transactions
        MAX                 // to mod with
        );

    cuda_ret = hipDeviceSynchronize();
    err_check(cuda_ret, (char*)"Unable to launch hash kernel!", 5);

    // Get the hash array from device memory
    unsigned int* hash_array = (unsigned int*)calloc(trials, sizeof(unsigned int));
    cuda_ret = hipMemcpy(hash_array, device_hash_array, trials * sizeof(unsigned int), hipMemcpyDeviceToHost);
    err_check(cuda_ret, (char*)"Unable to read hash from device memory!", 6);

    free(nonce_array);
    free(hash_array);
    free(transactions);
    hipFree(device_transactions);
    

    // ------ Step 3: Find the nonce with the minimum hash value ------ //
    // TODO Problem 2: find the minimum in the GPU by reduction
    
   // Allocate device memory for the minimum hash values and nonces
   unsigned int* d_min_hash;
   unsigned int* d_min_nonce;
   hipMalloc(&d_min_hash, (trials / (2 * BLOCK_SIZE)) * sizeof(unsigned int));
   hipMalloc(&d_min_nonce, (trials / (2 * BLOCK_SIZE)) * sizeof(unsigned int));

   
   // Set the block and grid dimensions for the reduction kernel
   dim3 dimBlock2(BLOCK_SIZE, 1, 1);
   dim3 dimGrid2((trials / (2 * BLOCK_SIZE)), 1, 1);

   // Launch the reduction kernel to find the local min hash values and local min nonce values
   reduction_kernel<<<dimGrid2, dimBlock2>>>(d_min_hash, d_min_nonce, device_hash_array, device_nonce_array, trials);

   // Copy the local min hash values and local min nonce values to the system memory
   unsigned int min_hash = MAX;
   unsigned int min_nonce = MAX;
   unsigned int* minHashArray = (unsigned int*)malloc((trials / (2 * BLOCK_SIZE)) * sizeof(unsigned int));
   unsigned int* minNonceArray = (unsigned int*)malloc((trials / (2 * BLOCK_SIZE)) * sizeof(unsigned int));
   hipMemcpy(minHashArray, d_min_hash, (trials / (2 * BLOCK_SIZE)) * sizeof(unsigned int), hipMemcpyDeviceToHost);
   hipMemcpy(minNonceArray, d_min_nonce, (trials / (2 * BLOCK_SIZE)) * sizeof(unsigned int), hipMemcpyDeviceToHost);

   // Find the global min hash value and min nonce value serially using the CPU

   for (int i = 0; i < (trials / (2 * BLOCK_SIZE)); i++) {
       if (minHashArray[i] < min_hash) {
           min_hash = minHashArray[i];
           min_nonce = minNonceArray[i];
       }
   }

    
    // Free memory
    hipFree(d_min_hash);
    hipFree(d_min_nonce);
    hipFree(device_hash_array);
    hipFree(device_nonce_array);
    free(minHashArray);
    free(minNonceArray);


    stopTime(&timer);
    // ----------------------------------------------------------------------------- //
    // -------- Finish Mining ------------------------------------------------------ //


    // Get if suceeded
    char* res = (char*)malloc(8 * sizeof(char));
    if (min_hash < TARGET)  res = (char*)"Success!";
    else                    res = (char*)"Failure.";

    // Show results in console
    if (DEBUG) 
        printf("%s\n   Min hash:  %u\n   Min nonce: %u\n   %f seconds\n",
            res,
            min_hash,
            min_nonce,
            elapsedTime(timer)
        );

    // Print results
    fprintf(output_file, "%s\n%u\n%u\n", res, min_hash, min_nonce);
    fprintf(time_file, "%f\n", elapsedTime(timer));

    // Cleanup
    fclose(time_file);
    fclose(output_file);

    return 0;
} // End Main -------------------------------------------- //



/* Generate Hash ----------------------------------------- //
*   Generates a hash value from a nonce and transaction list.
*/
unsigned int generate_hash(unsigned int nonce, unsigned int index, unsigned int* transactions, unsigned int n_transactions) {

    unsigned int hash = (nonce + transactions[0] * (index + 1)) % MAX;
    for (int j = 1; j < n_transactions; j++) {
        hash = (hash + transactions[j] * (index + 1)) % MAX;
    }
    return hash;

} // End Generate Hash ---------- //



/* Read File -------------------- //
*   Reads in a file of transactions. 
*/
void read_file(char* file, unsigned int* transactions, unsigned int n_transactions) {

    // open file
    FILE* trans_file = fopen(file, "r");
    if (trans_file == NULL)
        fprintf(stderr, "ERROR: could not read the transaction file.\n"),
        exit(-1);

    // read items
    char line[100] = { 0 };
    for (int i = 0; i < n_transactions && fgets(line, 100, trans_file); ++i) {
        char* p;
        transactions[i] = strtof(line, &p);
    }

    fclose(trans_file);

} // End Read File ------------- //



/* Error Check ----------------- //
*   Exits if there is a CUDA error.
*/
void err_check(hipError_t ret, char* msg, int exit_code) {
    if (ret != hipSuccess)
        fprintf(stderr, "%s \"%s\".\n", msg, hipGetErrorString(ret)),
        exit(exit_code);
} // End Error Check ----------- //
